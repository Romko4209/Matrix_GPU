#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include<ctime>

using namespace std;

#define BLOCK_SIZE  1024// ����� ���������  
#define N           1024// ����� �������



//�������� �������
__global__ void MatrixMul(float * a, float * b, int n, float * c)
{
	int   bx = blockIdx.x;     // block index
	int   by = blockIdx.y;
	int   tx = threadIdx.x;        // thread index
	int   ty = threadIdx.y;
	float sum = 0.0f;           // �����
	int   ia = n * BLOCK_SIZE * by + n * ty;   // a [i][0]
	int   ib = BLOCK_SIZE * bx + tx;

	// ��������
	for (int k = 0; k < n; k++)
		sum += a[ia + k] * b[ib + k*n];

	// �������� � ���� ���������
	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	c[ic + n * ty + tx] = sum;
}


int main(int argc, char *  argv[])
{


	int numBytes = N * N * sizeof(float);

	// �������� �����
	float * a = new float[N*N];
	float * b = new float[N*N];
	float * c = new float[N*N];

	// ���������� �������
	for (int i = 0; i < N*N; i++)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}


	// �������� �� ������ ����� GPU
	float * adev = NULL;
	float * bdev = NULL;
	float * cdev = NULL;

	//�������� ���'��� ��� ������ �� ��������
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);


	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	// ������� �����
	hipEventRecord(start, 0);

	// ������� ��� � ���'��� ���������
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);
	//����������

	// �������� �������
	MatrixMul << <blocks, threads >> > (adev, bdev, N, cdev);


	// ��������� ���������� � ����� �
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	// ����� �����
	hipEventRecord(stop, 0);


	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

	// ���������� �� ��������
	// ������� �����
	double begin = clock();

	for (int i = 0; i<N; i++) {
		for (int l = 0; l<N; l++) {
			float s = 0;
			for (int j = 0; j<N; j++)
				s += a[i*N + j] * b[j*N + l];

			c[i*N + l] = s;
		}
	}

	//����� �����
	double end = clock();
	double cpuTime = double(end - begin) / CLOCKS_PER_SEC*1000.0;
	printf("time spent executing by the CPU: %.10f milliseconds\n", cpuTime);


	////��������� �������
	//for (int i = 0; i < N*N; i++)
	//{

	//	if (i % N == 0)
	//		cout << "\n";
	//	cout << c[i] << " ";
	//}

	// ��������� �����
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);

	delete a;
	delete b;
	delete c;
	system("pause");
	return 0;
}